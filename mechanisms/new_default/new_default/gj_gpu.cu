#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace new_default_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto* _pp_var_g __attribute__((unused)) = params_.parameters[0];\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    unsigned lane_mask_ = arb::gpu::ballot(0xffffffff, tid_<n_);
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto peer_indexi_ = _pp_var_peer_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type v_peer = _pp_var_vec_v[peer_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type i = 0;
        i = _pp_var_g[tid_]*(v-v_peer);
        current_ = i;
        conductivity_ = _pp_var_g[tid_];
        ::arb::gpu::reduce_by_key(_pp_var_weight[tid_]*conductivity_,_pp_var_vec_g, node_indexi_, lane_mask_);
        ::arb::gpu::reduce_by_key(_pp_var_weight[tid_]*current_,_pp_var_vec_i, node_indexi_, lane_mask_);
    }
}

} // namespace

void mechanism_gj_gpu_init_(arb_mechanism_ppack* p) {}

void mechanism_gj_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_gj_gpu_advance_state_(arb_mechanism_ppack* p) {}

void mechanism_gj_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_gj_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_gj_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace new_default_catalogue
} // namespace arb
