#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace new_default_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_a0m __attribute__((unused)) = params_.globals[0];\
auto _pp_var_vhalfm __attribute__((unused)) = params_.globals[1];\
auto _pp_var_zetam __attribute__((unused)) = params_.globals[2];\
auto _pp_var_gmm __attribute__((unused)) = params_.globals[3];\
auto _pp_var_a0h __attribute__((unused)) = params_.globals[4];\
auto _pp_var_vhalfh __attribute__((unused)) = params_.globals[5];\
auto _pp_var_zetah __attribute__((unused)) = params_.globals[6];\
auto _pp_var_gmh __attribute__((unused)) = params_.globals[7];\
auto _pp_var_sha __attribute__((unused)) = params_.globals[8];\
auto _pp_var_shi __attribute__((unused)) = params_.globals[9];\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_h __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_v __attribute__((unused)) = params_.state_vars[2];\
auto* _pp_var_minf __attribute__((unused)) = params_.state_vars[3];\
auto* _pp_var_mtau __attribute__((unused)) = params_.state_vars[4];\
auto* _pp_var_hinf __attribute__((unused)) = params_.state_vars[5];\
auto* _pp_var_htau __attribute__((unused)) = params_.state_vars[6];\
auto* _pp_var_gbar __attribute__((unused)) = params_.parameters[0];\
auto* _pp_var_q10 __attribute__((unused)) = params_.parameters[1];\
auto& _pp_var_ion_k __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_k_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__device__
void trates(arb_mechanism_ppack params_, int tid_, arb_value_type v, arb_value_type celsius) {
    PPACK_IFACE_BLOCK;
    arb_value_type qt, ll0_, ll2_, ll1_, ll3_;
    ll3_ =  0.;
    ll2_ =  0.;
    ll1_ =  0.;
    ll0_ =  0.;
    qt = pow(_pp_var_q10[tid_], (celsius- 24.0)* 0.10000000000000001);
    _pp_var_minf[tid_] =  1.0/( 1.0+exp( -(v-_pp_var_sha- 7.5999999999999996)* 0.071428571428571425));
    ll0_ = exp(_pp_var_zetam*_pp_var_gmm*(v-_pp_var_vhalfm));
    ll1_ = exp(_pp_var_zetam*(v-_pp_var_vhalfm));
    _pp_var_mtau[tid_] = ll0_/(qt*_pp_var_a0m*( 1.0+ll1_));
    _pp_var_hinf[tid_] =  1.0/( 1.0+exp((v-_pp_var_shi+ 47.399999999999999)* 0.16666666666666666));
    ll2_ = exp(_pp_var_zetah*_pp_var_gmh*(v-_pp_var_vhalfh));
    ll3_ = exp(_pp_var_zetah*(v-_pp_var_vhalfh));
    _pp_var_htau[tid_] = ll2_/(qt*_pp_var_a0h*( 1.0+ll3_));
}

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        trates(params_, tid_, v, celsius);
        _pp_var_m[tid_] = _pp_var_minf[tid_];
        _pp_var_h[tid_] = _pp_var_hinf[tid_];
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type b_0_, a_0_, b_1_, ll0_, a_1_, ll2_, ll1_, ll3_;
        ll3_ =  0.;
        ll2_ =  0.;
        ll1_ =  0.;
        ll0_ =  0.;
        trates(params_, tid_, v, celsius);
        a_0_ = _pp_var_mtau[tid_];
        b_0_ = _pp_var_minf[tid_];
        ll0_ =  -dt/a_0_;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] = b_0_+(_pp_var_m[tid_]-b_0_)*ll1_;
        a_1_ = _pp_var_htau[tid_];
        b_1_ = _pp_var_hinf[tid_];
        ll2_ =  -dt/a_1_;
        ll3_ = ( 1.0+ 0.5*ll2_)/( 1.0- 0.5*ll2_);
        _pp_var_h[tid_] = b_1_+(_pp_var_h[tid_]-b_1_)*ll3_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_k_indexi_ = _pp_var_ion_k_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type ek = _pp_var_ion_k.reversal_potential[ion_k_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ik = 0;
        ik = _pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_]*(v-ek);
        current_ = ik;
        conductivity_ = _pp_var_gbar[tid_]*_pp_var_m[tid_]*_pp_var_h[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_k.current_density[ion_k_indexi_] = fma(10.0*_pp_var_weight[tid_], ik, _pp_var_ion_k.current_density[ion_k_indexi_]);
    }
}

} // namespace

void mechanism_kamt_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 2}, block_dim>>>(*p);
}

void mechanism_kamt_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_kamt_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_kamt_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_kamt_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_kamt_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace new_default_catalogue
} // namespace arb
