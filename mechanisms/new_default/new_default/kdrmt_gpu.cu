#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace new_default_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_a0m __attribute__((unused)) = params_.globals[0];\
auto _pp_var_zetam __attribute__((unused)) = params_.globals[1];\
auto _pp_var_gmm __attribute__((unused)) = params_.globals[2];\
auto _pp_var_alpm __attribute__((unused)) = params_.globals[3];\
auto _pp_var_betm __attribute__((unused)) = params_.globals[4];\
auto* _pp_var_m __attribute__((unused)) = params_.state_vars[0];\
auto* _pp_var_v __attribute__((unused)) = params_.state_vars[1];\
auto* _pp_var_minf __attribute__((unused)) = params_.state_vars[2];\
auto* _pp_var_mtau __attribute__((unused)) = params_.state_vars[3];\
auto* _pp_var_gbar __attribute__((unused)) = params_.parameters[0];\
auto* _pp_var_vhalfm __attribute__((unused)) = params_.parameters[1];\
auto* _pp_var_q10 __attribute__((unused)) = params_.parameters[2];\
auto& _pp_var_ion_k __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_k_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__device__
void trates(arb_mechanism_ppack params_, int tid_, arb_value_type v, arb_value_type celsius) {
    PPACK_IFACE_BLOCK;
    arb_value_type betm_t, tmp, alpm_t, qt;
    qt = pow(_pp_var_q10[tid_], (celsius- 24.0)* 0.10000000000000001);
    _pp_var_minf[tid_] =  1.0/( 1.0+exp( -(v- 21.0)* 0.10000000000000001));
    tmp = _pp_var_zetam*(v-_pp_var_vhalfm[tid_]);
    alpm_t = exp(tmp);
    betm_t = exp(_pp_var_gmm*tmp);
    _pp_var_mtau[tid_] = betm_t/(qt*_pp_var_a0m*( 1.0+alpm_t));
}

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        trates(params_, tid_, v, celsius);
        _pp_var_m[tid_] = _pp_var_minf[tid_];
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void advance_state(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type dt = _pp_var_vec_dt[node_indexi_];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type b_0_, a_0_, ll0_, ll1_;
        ll1_ =  0.;
        ll0_ =  0.;
        trates(params_, tid_, v, celsius);
        a_0_ = _pp_var_mtau[tid_];
        b_0_ = _pp_var_minf[tid_];
        ll0_ =  -dt/a_0_;
        ll1_ = ( 1.0+ 0.5*ll0_)/( 1.0- 0.5*ll0_);
        _pp_var_m[tid_] = b_0_+(_pp_var_m[tid_]-b_0_)*ll1_;
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_k_indexi_ = _pp_var_ion_k_index[tid_];
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type conductivity_ = 0;
        arb_value_type current_ = 0;
        arb_value_type ek = _pp_var_ion_k.reversal_potential[ion_k_indexi_];
        arb_value_type v = _pp_var_vec_v[node_indexi_];
        arb_value_type ik = 0;
        ik = _pp_var_gbar[tid_]*_pp_var_m[tid_]*(v-ek);
        current_ = ik;
        conductivity_ = _pp_var_gbar[tid_]*_pp_var_m[tid_];
        _pp_var_vec_g[node_indexi_] = fma(10.0*_pp_var_weight[tid_], conductivity_, _pp_var_vec_g[node_indexi_]);
        _pp_var_vec_i[node_indexi_] = fma(10.0*_pp_var_weight[tid_], current_, _pp_var_vec_i[node_indexi_]);
        _pp_var_ion_k.current_density[ion_k_indexi_] = fma(10.0*_pp_var_weight[tid_], ik, _pp_var_ion_k.current_density[ion_k_indexi_]);
    }
}

} // namespace

void mechanism_kdrmt_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 1}, block_dim>>>(*p);
}

void mechanism_kdrmt_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_kdrmt_gpu_advance_state_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    advance_state<<<grid_dim, block_dim>>>(*p);
}

void mechanism_kdrmt_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_kdrmt_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_kdrmt_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace new_default_catalogue
} // namespace arb
