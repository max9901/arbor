#include "hip/hip_runtime.h"
#include <arbor/gpu/gpu_common.hpp>
#include <arbor/gpu/math_cu.hpp>
#include <arbor/gpu/reduce_by_key.hpp>
#include <arbor/mechanism_abi.h>

namespace arb {
namespace new_default_catalogue {

#define PPACK_IFACE_BLOCK \
auto  _pp_var_width             __attribute__((unused)) = params_.width;\
auto  _pp_var_n_detectors       __attribute__((unused)) = params_.n_detectors;\
auto* _pp_var_vec_ci            __attribute__((unused)) = params_.vec_ci;\
auto* _pp_var_vec_di            __attribute__((unused)) = params_.vec_di;\
auto* _pp_var_vec_t             __attribute__((unused)) = params_.vec_t;\
auto* _pp_var_vec_dt            __attribute__((unused)) = params_.vec_dt;\
auto* _pp_var_vec_v             __attribute__((unused)) = params_.vec_v;\
auto* _pp_var_vec_i             __attribute__((unused)) = params_.vec_i;\
auto* _pp_var_vec_g             __attribute__((unused)) = params_.vec_g;\
auto* _pp_var_temperature_degC  __attribute__((unused)) = params_.temperature_degC;\
auto* _pp_var_diam_um           __attribute__((unused)) = params_.diam_um;\
auto* _pp_var_time_since_spike  __attribute__((unused)) = params_.time_since_spike;\
auto* _pp_var_node_index        __attribute__((unused)) = params_.node_index;\
auto* _pp_var_peer_index        __attribute__((unused)) = params_.peer_index;\
auto* _pp_var_multiplicity      __attribute__((unused)) = params_.multiplicity;\
auto* _pp_var_state_vars        __attribute__((unused)) = params_.state_vars;\
auto* _pp_var_weight            __attribute__((unused)) = params_.weight;\
auto& _pp_var_events            __attribute__((unused)) = params_.events;\
auto& _pp_var_mechanism_id      __attribute__((unused)) = params_.mechanism_id;\
auto& _pp_var_index_constraints __attribute__((unused)) = params_.index_constraints;\
auto _pp_var_R __attribute__((unused)) = params_.globals[0];\
auto _pp_var_F __attribute__((unused)) = params_.globals[1];\
auto* _pp_var_coeff __attribute__((unused)) = params_.state_vars[0];\
auto& _pp_var_ion_x __attribute__((unused)) = params_.ion_states[0];\
auto* _pp_var_ion_x_index __attribute__((unused)) = params_.ion_states[0].index;\
//End of IFACEBLOCK

namespace {

using ::arb::gpu::exprelr;
using ::arb::gpu::safeinv;
using ::arb::gpu::min;
using ::arb::gpu::max;

__global__
void init(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto node_indexi_ = _pp_var_node_index[tid_];
        arb_value_type zx = _pp_var_ion_x.ionic_charge[0];
        arb_value_type celsius = _pp_var_temperature_degC[node_indexi_];
        _pp_var_coeff[tid_] = _pp_var_R*(celsius+ 273.14999999999998)/(zx*_pp_var_F)* 1000.0;
    }
}

__global__
void multiply(arb_mechanism_ppack params_) {
    PPACK_IFACE_BLOCK;
    auto tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    auto idx_ = blockIdx.y;    if(tid_<_pp_var_width) {
        _pp_var_state_vars[idx_][tid_] *= _pp_var_multiplicity[tid_];
    }
}

__global__
void compute_currents(arb_mechanism_ppack params_) {
    int n_ = params_.width;
    int tid_ = threadIdx.x + blockDim.x*blockIdx.x;
    PPACK_IFACE_BLOCK;
    if (tid_<n_) {
        auto ion_x_indexi_ = _pp_var_ion_x_index[tid_];
        arb_value_type xi = _pp_var_ion_x.internal_concentration[ion_x_indexi_];
        arb_value_type xo = _pp_var_ion_x.external_concentration[ion_x_indexi_];
        arb_value_type ex = 0;
        ex = _pp_var_coeff[tid_]*log(xo/xi);
        _pp_var_ion_x.reversal_potential[ion_x_indexi_] = ex;
    }
}

} // namespace

void mechanism_nernst_gpu_init_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    init<<<grid_dim, block_dim>>>(*p);
    if (!p->multiplicity) return;
    multiply<<<dim3{grid_dim, 0}, block_dim>>>(*p);
}

void mechanism_nernst_gpu_compute_currents_(arb_mechanism_ppack* p) {
    auto n = p->width;
    unsigned block_dim = 128;
    unsigned grid_dim = ::arb::gpu::impl::block_count(n, block_dim);
    compute_currents<<<grid_dim, block_dim>>>(*p);
}

void mechanism_nernst_gpu_advance_state_(arb_mechanism_ppack* p) {}

void mechanism_nernst_gpu_write_ions_(arb_mechanism_ppack* p) {}

void mechanism_nernst_gpu_post_event_(arb_mechanism_ppack* p) {}
void mechanism_nernst_gpu_apply_events_(arb_mechanism_ppack* p, arb_deliverable_event_stream* events) {}

} // namespace new_default_catalogue
} // namespace arb
